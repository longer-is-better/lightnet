#include "tools_cuda.cuh"


void check_device_data(float* p_data, size_t ele) {
  VLOG(8) << "check_device_data";
  float t[ele];
  checkCudaErrors(hipMemcpy(t, p_data, ele * sizeof(float), hipMemcpyDeviceToHost));
  for (int i = 0; i < ele; i++) {
    VLOG(8) << t[i];
  }
}