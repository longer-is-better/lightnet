#include "hip/hip_runtime.h"
#include "operator_reduce.cuh"
#include "kernel_reduce.cuh"
#include "kernel_map.cuh"
#include "kernel_others.cuh"
#include "tools_common.cuh"

Reduce::Reduce(REDUCE_OP op):_reduce_op(op) {}

Reduce::Reduce(Tensor* A, REDUCE_OP op)
    : Operator({A}, {new Tensor()}), _reduce_op(op) {
  ;
}

std::string Reduce::type_str() { return std::string("Reduce"); }

Reduce* Reduce::copy() { return new Reduce(_reduce_op); }

void Reduce::infer_shape() {
    _output_tensors[0]->set_shape({});
}


void Reduce::forward() {

    dim3 BLOCK = 512;
    size_t shared_mem = BLOCK.x * sizeof(float);

    float *work_space;
    checkCudaErrors(hipMalloc(&work_space, _input_tensors[0]->_total_size));

    size_t work_space_elecount = _input_tensors[0]->_element_count;
    while (work_space_elecount != 1){
        dim3 GRID = ceil(work_space_elecount, BLOCK.x * 2) / (BLOCK.x * 2);
        kreduce<<<GRID, BLOCK, shared_mem, _cudastream>>>(
            _input_tensors[0]->_element_count,
            work_space_elecount,
            work_space,
            work_space,
            REDUCE_OP::SUM
        );
        work_space_elecount = GRID.x;
    }
    checkCudaErrors(hipMemcpy(_output_tensors[0]->_p_data, work_space, sizeof(float), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipFree(work_space));
}


void Reduce::backward() {
    dim3 BLOCK;
    dim3 GRID;
    size_t shared_mem;

    BLOCK = dim3(32);
    GRID = dim3(ceil(_input_tensors[0]->_element_count, 32) / 32);
    shared_mem = 0;

    float alpha;
    switch (_reduce_op) {
        case REDUCE_OP::SUM:
            alpha = 1.f;
            break;
        case REDUCE_OP::AVG:
            alpha = 1.f / _input_tensors[0]->_element_count;
            break;
        
        default:
            break;
    }
    // kmemset_d<<<GRID, BLOCK, shared_mem, _cudastream>>>(
    //     _input_tensors[0]->_element_count,
    //     _input_tensors[0]->_p_gradient,
    //     alpha,
    //     _output_tensors[0]->_p_gradient
    // );
    checkCudaErrors(hipDeviceSynchronize());
    float sss[2];
    checkCudaErrors(hipMemcpy(sss, _input_tensors[0]->_p_gradient, _input_tensors[0]->_total_size, hipMemcpyDeviceToHost));




    Tensor s = _input_tensors[0]->grad();
    s.to(hipMemoryTypeHost);
    // D(VLOG(7) << _name << _reduce_op << " backward get input tensor[0] grad:" << s);
}