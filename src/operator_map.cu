#include "operator_map.cuh"
#include "kernel_map.cuh"
#include "kernel_elementwise.cuh"
#include "kernel_others.cuh"
#include "tools_common.cuh"



Map::Map(
    Tensor* A,
    MAP_OP op,
    float operand
):
    Operator({A}, {new Tensor()}),
    _operand(operand)
{
    ;
}

Operator *Map::copy()
{
    return new Map();
}

void Map::infer_shape() {
    _output_tensors[0]->set_shape(_input_tensors[0]->_shape);
}


void Map::forward() {
    dim3 BLOCK;
    dim3 GRID;
    size_t shared_mem;

    
    BLOCK = dim3(32);
    GRID = dim3((_input_tensors[0]->_element_count + BLOCK.x - 1) / BLOCK.x);
    shared_mem = 0;

    kmap<<<GRID, BLOCK, shared_mem, _cudastream>>>(
        _input_tensors[0]->_element_count,
        _input_tensors[0]->_p_data,
        _output_tensors[0]->_p_data,
        _map_op,
        _operand
    );
    checkCudaErrors(hipDeviceSynchronize());

}


void Map::backward() {
    dim3 BLOCK;
    dim3 GRID;
    size_t shared_mem;

    
    BLOCK = dim3(32);
    GRID = dim3((_input_tensors[0]->_element_count + BLOCK.x - 1) / BLOCK.x);
    shared_mem = 0;

    switch (_map_op) {
        case MAP_OP::ADD:
            kmemset<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _input_tensors[0]->_p_gradient,
                1.f
            );
            break;
        case MAP_OP::MULTIPLY:
            kmemset<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _input_tensors[0]->_p_gradient,
                _operand
            );
            break;
        case MAP_OP::POW:
            kmap<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _input_tensors[0]->_p_data,
                _input_tensors[0]->_p_gradient,
                MAP_OP::POW,
                _operand - 1
            );
            kmap_inplace<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _input_tensors[0]->_p_gradient,
                MAP_OP::MULTIPLY,
                _operand
            );
            break;
        case MAP_OP::ABS:
            kmap<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _input_tensors[0]->_p_data,
                _input_tensors[0]->_p_gradient,
                MAP_OP::SIGN,
                0.f
            );
            kelementwise_inplace<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _input_tensors[0]->_p_gradient,
                1.f,
                _output_tensors[0]->_p_gradient,
                ELE_OP::MULTIPLY
            );
            break;
        case MAP_OP::LOG:
            kelementwise<<<GRID, BLOCK, shared_mem, _cudastream>>>(
                _input_tensors[0]->_element_count,
                _output_tensors[0]->_p_gradient,
                1.f,
                _input_tensors[0]->_p_data,
                _input_tensors[0]->_p_gradient,
                ELE_OP::DIVIDE
            );
            break;
        
        default:
            break;
    }
}