#include "operator_elementwise.cuh"
#include "kernel_others.cuh"
#include "kernel_map.cuh"
#include "kernel_elementwise.cuh"


ElementWise::ElementWise(
    Tensor* A,
    Tensor* B,
    ELE_OP op
):
    Operator({A, B}, {new Tensor()})
{
    ;
}

Operator *ElementWise::copy()
{
    return new ElementWise();
}

void ElementWise::infer_shape() {
    CHECK_EQ(_input_tensors.size(), 2);
    CHECK_EQ(_input_tensors[0]->_dim_n, _input_tensors[1]->_dim_n);
    for (size_t i = 0; i < _input_tensors[0]->_dim_n; i++) {
        CHECK_EQ(_input_tensors[0]->_shape[i], _input_tensors[1]->_shape[i]);
    }
    CHECK_STREQ(_input_tensors[0]->_layout.c_str(), _input_tensors[1]->_layout.c_str());
    _output_tensors[0]->set_shape(_input_tensors[0]->_shape);
}


void ElementWise::forward() {
    dim3 BLOCK(32);
    dim3 GRID((_input_tensors[0]->_element_count + BLOCK.x - 1) / BLOCK.x);
    size_t shared_mem = 0;
    Tensor show;
    show = *_input_tensors[0];
    std::cout << "0:\n" << show;
    show = *_input_tensors[1];
    std::cout << "1:\n" << show;
    kelementwise<<<GRID, BLOCK, 0, _cudastream>>>(
        _input_tensors[0]->_element_count,
        _input_tensors[0]->_p_data,
        1.f,
        _input_tensors[1]->_p_data,
        _output_tensors[0]->_p_data,
        _ele_op
    );
    checkCudaErrors(hipDeviceSynchronize());
}


void ElementWise::backward() {
    dim3 BLOCK(32);
    dim3 GRID((_input_tensors[0]->_element_count + BLOCK.x - 1) / BLOCK.x);
    size_t shared_mem = 0;
    switch (_ele_op) {
        case ELE_OP::ADD:
            checkCudaErrors(hipMemcpyAsync(_input_tensors[0]->_p_gradient, _output_tensors[0]->_p_gradient, _output_tensors[0]->_total_size, hipMemcpyDeviceToDevice, _cudastream));
            checkCudaErrors(hipMemcpyAsync(_input_tensors[1]->_p_gradient, _output_tensors[0]->_p_gradient, _output_tensors[0]->_total_size, hipMemcpyDeviceToDevice, _cudastream));
            break;
        case ELE_OP::SUB:
            checkCudaErrors(hipMemcpyAsync(_input_tensors[0]->_p_gradient, _output_tensors[0]->_p_gradient, _output_tensors[0]->_total_size, hipMemcpyDeviceToDevice, _cudastream));
            kmap<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _output_tensors[0]->_p_gradient,
                _input_tensors[1]->_p_gradient,
                MAP_OP::MULTIPLY,
                -1.f
            );
            break;
        case ELE_OP::MULTIPLY:
            kelementwise<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _input_tensors[0]->_p_data,
                1.f,
                _output_tensors[0]->_p_gradient,
                _input_tensors[1]->_p_gradient,
                ELE_OP::MULTIPLY
            );
            kelementwise<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _input_tensors[1]->_p_data,
                1.f,
                _output_tensors[0]->_p_gradient,
                _input_tensors[0]->_p_gradient,
                ELE_OP::MULTIPLY
            );
            break;
        case ELE_OP::DIVIDE:
            kelementwise<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _output_tensors[0]->_p_gradient,
                1.f,
                _input_tensors[1]->_p_data,
                _input_tensors[0]->_p_gradient,
                ELE_OP::DIVIDE
            );
            kmap<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _input_tensors[1]->_p_data,
                _input_tensors[1]->_p_gradient,
                MAP_OP::POW,
                -2.f
            );
            kmap_inplace<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _input_tensors[1]->_p_gradient,
                MAP_OP::MULTIPLY,
                -1.f
            );
            kelementwise_inplace<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _input_tensors[1]->_p_gradient,
                1.f,
                _input_tensors[0]->_p_data,
                ELE_OP::MULTIPLY
            );
            kelementwise_inplace<<<GRID, BLOCK, 0, _cudastream>>>(
                _output_tensors[0]->_total_size,
                _input_tensors[1]->_p_gradient,
                1.f,
                _output_tensors[0]->_p_gradient,
                ELE_OP::MULTIPLY
            );
            break;

        default:
            break;
    }
}