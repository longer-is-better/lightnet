#include "hip/hip_runtime.h"
#include <random>
#include <chrono>
#include <string>
#include <hipblas.h>
#include <glog/logging.h>
#include <gtest/gtest.h>

#include "cnpy.cuh"
#include "tensor.cuh"
#include "tools_cuda.cuh"
#include "tools_common.cuh"
#include "kernel_bev_pool_v2.cuh"


struct DIM
{
    int x = 1;
    int y = 1;
    int z = 1;
    int w = 1;
    DIM(int inx, int iny, int inz, int inw) : x(inx), y(iny), z(inz), w(inw){};
    DIM(int inx, int iny, int inz) : x(inx), y(iny), z(inz){};
    DIM(int inx, int iny) : x(inx), y(iny){};
    DIM(int inx) : x(inx){};
    DIM(){};
    template <typename T>
    size_t size()
    {
        return x * y * z * w * sizeof(T);
    }
    size_t nums()
    {
        return x * y * z * w;
    }
};


template<
    typename TENSORTYPE,
    typename ACCTYPE,
    const int TC,
    const int TN,
    const int BC,
    const int BN
>
class test_bev_pool_v2_fma_tnc:
    public testing::TestWithParam<
        std::tuple<
            INPUT_FILE_TYPE,
            std::string,  // input files dir
            DIM,  // depth_shape
            DIM,  // feat_shape
            DIM  // out_shape
        >
    >
{
public:
    INPUT_FILE_TYPE inputfiletype;
    std::string input_files_dir;
    DIM depth_shape;
    DIM feat_shape;
    DIM out_shape;


    cnpy::NpyArray  ranks_bev,\
                    ranks_depth,\
                    ranks_feat,\
                    interval_lengths,\
                    interval_starts;

    int c, n_intervals,\
        *ranks_depth_host, *ranks_depth_device,\
        *ranks_feat_host, *ranks_feat_device,\
        *ranks_bev_host, *ranks_bev_device,\
        *ranks_bev_mask_host, *ranks_bev_mask_device,\
        *interval_starts_host, *interval_starts_device,\
        *interval_lengths_host, *interval_lengths_device,\
        *interval_starts_e_host, *interval_starts_e_device,\
        *interval_lengths_e_host, *interval_lengths_e_device;
    TENSORTYPE  *depth_host, *depth_device,\
                *feat_host, *feat_device,\
                *out_gt_host, *out_gt_device,\
                *out_test_host, *out_test_device;

    // hipStream_t stream;
    test_bev_pool_v2_fma_tnc();
    ~test_bev_pool_v2_fma_tnc();
};

template<
    typename TENSORTYPE,
    typename ACCTYPE,
    const int TC,
    const int TN,
    const int BC,
    const int BN
>
test_bev_pool_v2_fma_tnc<
    TENSORTYPE,
    ACCTYPE,
    TC,
    TN,
    BC,
    BN
>::test_bev_pool_v2_fma_tnc() {
    std::tie(
        inputfiletype,
        input_files_dir,
        depth_shape,
        feat_shape,
        out_shape
    ) = GetParam();

    if (inputfiletype == INPUT_FILE_TYPE::npy) {
        ranks_bev = cnpy::npy_load(input_files_dir + "/ranks_bev.npz.npy");
        ranks_depth = cnpy::npy_load(input_files_dir + "/ranks_depth.npz.npy");
        ranks_feat = cnpy::npy_load(input_files_dir + "/ranks_feat.npz.npy");
        interval_lengths = cnpy::npy_load(input_files_dir + "/interval_lengths.npz.npy");
        interval_starts = cnpy::npy_load(input_files_dir + "/interval_starts.npz.npy");
    }

    // checkCudaErrors(hipStreamCreate(&stream));

    depth_host = (TENSORTYPE*)malloc(depth_shape.size<TENSORTYPE>());
    feat_host = (TENSORTYPE*)malloc(feat_shape.size<TENSORTYPE>());
    ranks_depth_host = (int*)malloc(ranks_depth.num_bytes());
    ranks_feat_host = (int*)malloc(ranks_feat.num_bytes());
    ranks_bev_host = (int*)malloc(ranks_bev.num_bytes());
    ranks_bev_mask_host = (int*)malloc(interval_starts.num_bytes()); memset(ranks_bev_mask_host, 0, interval_starts.num_bytes());
    interval_starts_host = (int*)malloc(interval_starts.num_bytes());
    interval_lengths_host = (int*)malloc(interval_lengths.num_bytes());
    interval_starts_e_host = (int*)malloc(interval_starts.num_bytes());
    interval_lengths_e_host = (int*)malloc(interval_lengths.num_bytes());
    out_gt_host = (TENSORTYPE*)malloc(out_shape.size<TENSORTYPE>());
    out_test_host = (TENSORTYPE*)malloc(out_shape.size<TENSORTYPE>());


    c = feat_shape.w;
    n_intervals = 192 * 256;
    for(int i = 0; i < ranks_depth.num_vals; i++) ranks_depth_host[i] = (int)ranks_depth.data<float>()[i];
    for(int i = 0; i < ranks_feat.num_vals; i++) ranks_feat_host[i] = (int)ranks_feat.data<float>()[i];
    for(int i = 0; i < ranks_bev.num_vals; i++) ranks_bev_host[i] = (int)ranks_bev.data<float>()[i];
    for(int i = 0; i < interval_starts.num_vals; i++) interval_starts_host[i] = (int)interval_starts.data<float>()[i];
    for(int i = 0; i < interval_lengths.num_vals; i++) interval_lengths_host[i] = (int)interval_lengths.data<float>()[i];

    for (int i = 0; i < ranks_bev.num_vals; i++) {
        int idx = ranks_bev_host[i];
        if (idx != -1 && ranks_bev_mask_host[idx] == 0)
        ranks_bev_mask_host[idx] = 1;
    }
    int j = 0;
    for (int i = 0; i < interval_starts.num_vals; i++) {
        if (ranks_bev_mask_host[i] == 0) {
            interval_starts_e_host[i] = 0;
            interval_lengths_e_host[i] = 0;
        } else {
            interval_starts_e_host[i] = interval_starts_host[j];
            interval_lengths_e_host[i] = interval_lengths_host[j];
            j++;
        }
    }


    checkCudaErrors(hipMalloc(&depth_device, depth_shape.size<TENSORTYPE>()));
    checkCudaErrors(hipMalloc(&feat_device, feat_shape.size<TENSORTYPE>()));
    checkCudaErrors(hipMalloc(&ranks_depth_device, ranks_depth.num_bytes()));
    checkCudaErrors(hipMalloc(&ranks_feat_device, ranks_feat.num_bytes()));
    checkCudaErrors(hipMalloc(&ranks_bev_device, ranks_bev.num_bytes()));
    checkCudaErrors(hipMalloc(&ranks_bev_mask_device, interval_starts.num_bytes()));
    checkCudaErrors(hipMalloc(&interval_starts_device, interval_starts.num_bytes()));
    checkCudaErrors(hipMalloc(&interval_lengths_device, interval_lengths.num_bytes()));
    checkCudaErrors(hipMalloc(&interval_starts_e_device, interval_starts.num_bytes()));
    checkCudaErrors(hipMalloc(&interval_lengths_e_device, interval_lengths.num_bytes()));
    checkCudaErrors(hipMalloc(&out_gt_device, out_shape.size<TENSORTYPE>()));
    checkCudaErrors(hipMalloc(&out_test_device, out_shape.size<TENSORTYPE>()));

    // init data
    for (int i = 0; i < depth_shape.nums(); i++)
    {
        depth_host[i] = TENSORTYPE((i % 100) * 0.01);
    }
    for (int i = 0; i < feat_shape.nums(); i++)
    {
        feat_host[i] = TENSORTYPE((100 - (i % 100)) * 0.01);
    }


    checkCudaErrors(hipMemcpy(depth_device, depth_host, depth_shape.size<TENSORTYPE>(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(feat_device, feat_host, feat_shape.size<TENSORTYPE>(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ranks_depth_device, ranks_depth_host, ranks_depth.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ranks_feat_device, ranks_feat_host, ranks_feat.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ranks_bev_device, ranks_bev_host, ranks_bev.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(ranks_bev_mask_device, ranks_bev_mask_host, interval_starts.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(interval_starts_device, interval_starts_host, interval_starts.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(interval_lengths_device, interval_lengths_host, interval_lengths.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(interval_starts_e_device, interval_starts_e_host, interval_starts.num_bytes(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(interval_lengths_e_device, interval_lengths_e_host, interval_lengths.num_bytes(), hipMemcpyHostToDevice));





}

template<
    typename TENSORTYPE,
    typename ACCTYPE,
    const int TC,
    const int TN,
    const int BC,
    const int BN
>
test_bev_pool_v2_fma_tnc<
    TENSORTYPE,
    ACCTYPE,
    TC,
    TN,
    BC,
    BN
>::~test_bev_pool_v2_fma_tnc() {
    free(ranks_depth_host);
    free(ranks_feat_host);
    free(ranks_bev_host);
    free(ranks_bev_mask_host);
    free(interval_starts_host);
    free(interval_lengths_host);
    free(interval_starts_e_host);
    free(interval_lengths_e_host);
    free(depth_host);
    free(feat_host);
    free(out_gt_host);
    free(out_test_host);
    checkCudaErrors(hipFree(ranks_depth_device));
    checkCudaErrors(hipFree(ranks_feat_device));
    checkCudaErrors(hipFree(ranks_bev_device));
    checkCudaErrors(hipFree(ranks_bev_mask_device));
    checkCudaErrors(hipFree(interval_starts_device));
    checkCudaErrors(hipFree(interval_lengths_device));
    checkCudaErrors(hipFree(interval_starts_e_device));
    checkCudaErrors(hipFree(interval_lengths_e_device));
    checkCudaErrors(hipFree(depth_device));
    checkCudaErrors(hipFree(feat_device));
    checkCudaErrors(hipFree(out_gt_device));
    checkCudaErrors(hipFree(out_test_device));

    // checkCudaErrors(hipStreamDestroy(stream));
}

using test_bev_pool_v2_fma_tnc_ff_1_1_32_8 = \
    test_bev_pool_v2_fma_tnc<float, float, 1, 1, 32, 8>;

INSTANTIATE_TEST_SUITE_P(
    design,
    test_bev_pool_v2_fma_tnc_ff_1_1_32_8,
    testing::Combine(
        testing::Values(
            INPUT_FILE_TYPE::npy
        ),
        testing::Values(
            "/home/jovyan/lightnet/tests/test_kernels/test_bev_pool_v2_inputs/npy"
            // "/home/dongwei/Workspace/lightnet/tests/test_kernels/test_bev_pool_v2_inputs/npy"
        ),
        testing::Values(
            DIM(7, 120, 64, 120)
        ),
        testing::Values(
            DIM(7, 64, 120, 128)
        ),
        testing::Values(
            DIM(1, 192, 256, 128)
        )
    )
);

TEST_P(test_bev_pool_v2_fma_tnc_ff_1_1_32_8, 0){
    constexpr int TC = 2;
    constexpr int TN = 1;
    constexpr int BC = 64;
    constexpr int BN = 8;
    using TENSORTYPE = float;
    using ACCTYPE = float;




    GPU_TICK("bev_pool_v2_b256", hipStreamDefault);
    bev_pool_v2_b256(
        c,
        n_intervals,
        depth_device,
        feat_device,
        ranks_depth_device,
        ranks_feat_device,
        ranks_bev_device,
        interval_starts_device,
        interval_lengths_device,
        out_gt_device
    );
    GPU_TOCK("bev_pool_v2_b256", hipStreamDefault);
    std::cout << "bev_pool_v2_b256 cost: " << GPU_TICKTOCKS["bev_pool_v2_b256"].interval << " ms." << std::endl;
    checkCudaErrors(hipMemcpy(out_gt_host, out_gt_device, out_shape.size<float>(), hipMemcpyDeviceToHost));


    // GPU_TICK("bev_pool_v2_mz", hipStreamDefault);
    // // for (int i = 0; i < 100; i++) {
    //     bev_pool_v2_mz(
    //         interval_starts_host[n_intervals - 1] + interval_lengths_host[n_intervals - 1],
    //         c,
    //         n_intervals,
    //         depth_device,
    //         feat_device,
    //         ranks_depth_device,
    //         ranks_feat_device,
    //         ranks_bev_device,
    //         interval_starts_device,
    //         interval_lengths_device,
    //         out_test_device
    //     );
    // // }
    // GPU_TOCK("bev_pool_v2_mz", hipStreamDefault);
    // std::cout << "bev_pool_v2_mz cost: " << GPU_TICKTOCKS["bev_pool_v2_mz"].interval << " ms." << std::endl;
    // checkCudaErrors(hipMemcpy(out_test_host, out_test_device, out_shape.size<float>(), hipMemcpyDeviceToHost));



    GPU_TICK("bev_pool_v2_morethread", hipStreamDefault);
    // for (int i = 0; i < 100; i++) {
        bev_pool_v2_morethread(
            interval_starts_host[n_intervals - 1] + interval_lengths_host[n_intervals - 1],
            c,
            n_intervals,
            depth_device,
            feat_device,
            ranks_depth_device,
            ranks_feat_device,
            ranks_bev_device,
            interval_starts_device,
            interval_lengths_device,
            out_test_device
        );
    // }
    GPU_TOCK("bev_pool_v2_morethread", hipStreamDefault);
    std::cout << "bev_pool_v2_morethread cost: " << GPU_TICKTOCKS["bev_pool_v2_morethread"].interval << " ms." << std::endl;
    checkCudaErrors(hipMemcpy(out_test_host, out_test_device, out_shape.size<float>(), hipMemcpyDeviceToHost));






    // dim3 gridSize(
    //     (c + TC * BC - 1)/(TC * BC),
    //     (n_intervals + TN * BN - 1)/(TN * BN)
    // );
    // dim3 blockSize(BC, BN);
    // GPU_TICK("bev_pool_kernel", hipStreamDefault);
    // bev_pool_kernel<TENSORTYPE, ACCTYPE, TC, TN><<<gridSize, blockSize>>>(
    //     c, n_intervals,
    //     const_cast<const TENSORTYPE*>(depth_device),
    //     const_cast<const TENSORTYPE*>(feat_device),
    //     const_cast<const int*>(ranks_depth_device),
    //     const_cast<const int*>(ranks_feat_device),
    //     const_cast<const int*>(interval_starts_e_device),
    //     const_cast<const int*>(interval_lengths_e_device),
    //     out_test_device
    // );
    // GPU_TOCK("bev_pool_kernel", hipStreamDefault);
    // std::cout << "bev_pool_kernel cost: " << GPU_TICKTOCKS["bev_pool_kernel"].interval << " ms." << std::endl;
    // checkCudaErrors(hipMemcpy(out_test_host, out_test_device, out_shape.size<float>(), hipMemcpyDeviceToHost));


    for (int h = 0; h < out_shape.y; h++) {  // 192
        for (int w = 0; w < out_shape.z; w++) {  // 256
            for (int c = 0; c < out_shape.w; c++) {  // 128
                float out_gt_host_i = out_gt_host[
                    h * out_shape.z * out_shape.w + \
                    w * out_shape.w + \
                    c
                ] + 1;
                float out_test_host_i = out_test_host[
                    h * out_shape.z * out_shape.w + \
                    w * out_shape.w + \
                    c
                ] + 1;

                std::cout << "check pos[" << h << ", " << w << ", " << c << "]" << std::endl;
                std::cout << "out_gt_host_i: " <<  out_gt_host_i << "\nout_test_host_i: " <<  out_test_host_i << std::endl;

                ASSERT_LE(
                    abs(
                        (out_gt_host_i - out_test_host_i) / out_gt_host_i
                    ),
                    0.00001
                )   << "\npos[" << h << ", " << w << ", " << c << "]:"
                    << "\nout_gt_host_i: " <<  out_gt_host_i
                    << "\nout_test_host_i: " <<  out_test_host_i
                    << std::endl << std::endl;
            }
            break;
        }
    }




}