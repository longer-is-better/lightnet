#include "hip/hip_runtime.h"
#include <random>
#include <hipblas.h>
#include <glog/logging.h>
#include <gtest/gtest.h>

#include "tensor.cuh"
#include "tools_cuda.cuh"
#include "tools_common.cuh"
#include "kernel_matmul.cuh"

class test_matmul:
    public testing::TestWithParam<
        std::tuple<
            bool,  // trans W
            bool,  // trans X
            int,  // m
            int,  // n
            int,  // k
            std::function<float(const std::vector<int>&)>,  // W gen
            std::function<float(const std::vector<int>&)>,  // X gen
            dim3  // block
        >
    >
{
public:
    bool trans_W, trans_X;
    int m ,n, k;
    std::function<float(const std::vector<int>&)> W_gen, X_gen;
    dim3 BLOCK;
    
    float alpha = 1.f, beta = 0.f;
    size_t W_size, X_size, Y_size;
    float *W_host, *X_host, *Y_ground_truth_host, *Y_predict_host, *W_device, *X_device, *Y_ground_truth_device, *Y_predict_device;
    dim3 GRID;
    size_t shared_mem;

    hipblasHandle_t handle = nullptr;

    test_matmul();
    ~test_matmul();
};

test_matmul::test_matmul() {
    std::tie(
        trans_W,
        trans_X,
        m,
        n,
        k,
        W_gen,
        X_gen,
        BLOCK
    ) = GetParam();

    hipblasCreate(&handle);
    W_size = m * k * sizeof(float);
    X_size = k * n * sizeof(float);
    Y_size = m * n * sizeof(float);

    CHECK_NOTNULL(W_host = (float*)malloc(W_size));
    CHECK_NOTNULL(X_host = (float*)malloc(X_size));
    CHECK_NOTNULL(Y_ground_truth_host = (float*)malloc(Y_size));
    CHECK_NOTNULL(Y_predict_host = (float*)malloc(Y_size));
    checkCudaErrors(hipMalloc(&W_device, W_size));
    checkCudaErrors(hipMalloc(&X_device, X_size));
    checkCudaErrors(hipMalloc(&Y_ground_truth_device, Y_size));
    checkCudaErrors(hipMalloc(&Y_predict_device, Y_size));

    for (int r = 0; r < m; r++) for (int c = 0; c < k; c++) W_host[r * k + c] = W_gen({r, c});
    for (int r = 0; r < k; r++) for (int c = 0; c < n; c++) X_host[r * n + c] = X_gen({r, c});
    checkCudaErrors(hipMemcpy(W_device, W_host, W_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(X_device, X_host, X_size, hipMemcpyHostToDevice));


    GRID = dim3(ceil(n, BLOCK.x)/BLOCK.x, ceil(m, BLOCK.y)/BLOCK.y);
    shared_mem = BLOCK.x * BLOCK.y * sizeof(float) * 2;

}

test_matmul::~test_matmul() {
    free(W_host);
    free(X_host);
    free(Y_ground_truth_host);
    free(Y_predict_host);
    checkCudaErrors(hipFree(W_device));
    checkCudaErrors(hipFree(X_device));
    checkCudaErrors(hipFree(Y_ground_truth_device));
    checkCudaErrors(hipFree(Y_predict_device));

    hipblasDestroy(handle);
}


INSTANTIATE_TEST_SUITE_P(
    design,
    test_matmul,
    testing::Values(
        std::make_tuple(
            false,
            true,
            1,
            1,
            1,
            get_rand_data_gen<float, std::uniform_real_distribution>(1.f, 1.f),
            get_rand_data_gen<float, std::uniform_real_distribution>(2.f, 2.f),
            dim3(16, 16)
        )
    )
);


INSTANTIATE_TEST_SUITE_P(
    exhaustive_combine,
    test_matmul,
    testing::Combine(
        testing::Values(true, false),
        testing::Values(true, false),
        testing::Values(1, 8, 64, 512),
        testing::Values(1, 128),
        testing::Values(1, 256, 1023),
        testing::Values(
            get_rand_data_gen<float, std::uniform_real_distribution>(-1.f, 1.f)
        ),
        testing::Values(
            get_rand_data_gen<float, std::uniform_real_distribution>(-1.f, 1.f)
        ),
        testing::Values(
            dim3(2, 2),
            dim3(8, 8),
            dim3(32, 32)
        )
    )
);

TEST_P(test_matmul, positive){
    std::vector<size_t> W_shape = trans_W ? std::vector<size_t>{size_t(k), size_t(m)} : std::vector<size_t>{size_t(m), size_t(k)};
    Tensor show_W(W_shape, hipMemoryTypeDevice, W_device);
    VLOG(8) << "show W \n" << show_W;

    std::vector<size_t> X_shape = trans_X ? std::vector<size_t>{size_t(n), size_t(k)} : std::vector<size_t>{size_t(k), size_t(n)};
    Tensor show_X(X_shape, hipMemoryTypeDevice, X_device);
    VLOG(8) << "show X \n" << show_X;

    hipblasSgemm(
        handle,
        trans_X ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        trans_W ? HIPBLAS_OP_T : HIPBLAS_OP_N,
        n,
        m,
        k,
        &alpha,
        X_device,
        trans_X ? k : n,
        W_device,
        trans_W ? m : k,
        &beta,
        Y_ground_truth_device,
        n
    );
    hipMemcpy(Y_ground_truth_host, Y_ground_truth_device, Y_size, hipMemcpyDeviceToHost);

    Tensor gt({size_t(m), size_t(n)}, hipMemoryTypeHost, Y_ground_truth_host);
    VLOG(8) << "show gt \n" << gt;

    kmatmul<<<GRID, BLOCK, shared_mem, hipStreamDefault>>>(
        trans_W,
        trans_X,
        m,
        k,
        n,
        W_device,
        X_device,
        Y_predict_device
    );
    checkCudaErrors(hipStreamSynchronize(hipStreamDefault));
    hipMemcpy(Y_predict_host, Y_predict_device, Y_size, hipMemcpyDeviceToHost);

    Tensor pd({size_t(m), size_t(n)}, hipMemoryTypeHost, Y_predict_host);
    VLOG(8) << "show pd \n" << pd;

    for (int r = 0; r < m; r++) {
        for (int c = 0; c < n; c++) {
            ASSERT_NEAR(
                Y_predict_host[r * n + c],
                Y_ground_truth_host[r * n + c],
                0.00005
            ) << "\ntrans_W: " + std::to_string(trans_W) +\
                 "\ntrans_X: " + std::to_string(trans_X) +\
                 "\nm: " + std::to_string(m) +\
                 "\nn: " + std::to_string(n) +\
                 "\nk: " + std::to_string(k) +\
                 "\nBLOCK: " << BLOCK\
                 << "at [" << std::to_string(r) << ", " << std::to_string(c) << "]";
        }
    }
}