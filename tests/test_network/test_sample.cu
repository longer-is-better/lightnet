#include <random>
#include <gtest/gtest.h>

#include "compute_graph.cuh"
#include "network.cuh"

#include "compute_graph_l1loss.cuh"
#include "operators.cuh"

#include "operator_elementwise.cuh"

#include "tools_cuda.cuh"


TEST(network, smoke) {
    ComputeGraph test_graph;
    test_graph._input_tensors.push_back(new Tensor());

    test_graph._weight_tensors.push_back(new Tensor({2, 2}));
    for (int i = 0; i < test_graph._weight_tensors[0]->_element_count; i++) {
        test_graph._weight_tensors[0]->_p_data[i] = i;
    }
    Operator *ele = new ElementWise(test_graph._input_tensors[0], test_graph._weight_tensors[0], ELE_OP::ADD);

    Network test_net(&test_graph, true, hipStreamDefault);
    test_net.to(hipMemoryTypeDevice);

    std::vector<Tensor*> sample_inputs{new Tensor({2, 2})};
    sample_inputs[0]->fill_data_random(0.9, 1.0);
    test_net.init(sample_inputs, "");

    test_net._weight_tensors[0]->update_weights(1.f, hipStreamDefault);
    for (int i = 0; i < 2; i++){
        Tensor t1({2, 2});
        t1.malloc_gradient();
        for (int i = 0; i < t1._element_count; i++) {
            t1._p_data[i] = i/10.f;
        }
        auto outs = test_net.forward({&t1});
        for (auto o: outs) {
            Tensor o_h(*o);
            o_h.to(hipMemoryTypeHost);
            std::cout << o_h;
        }
        checkCudaErrors(hipMemcpy(test_net.get_output_tensors()[0]->_p_gradient, sample_inputs[0]->_p_data, 16, hipMemcpyHostToDevice));
        
        test_net.backward();
        test_net.update_weights(0.5);
        Tensor w(*test_net._weight_tensors[0]);
        w.to(hipMemoryTypeHost);
        std::cout << "weight: " << w << std::endl;


        std::cout << "---------------" << std::endl;
    }
}

TEST(network, mm) {
    ComputeGraph *mm_graph = new ComputeGraph();
    mm_graph->_input_tensors.push_back(new Tensor());
    mm_graph->_weight_tensors.push_back(new Tensor({2, 2}));
    new MatMul(mm_graph->_weight_tensors[0], mm_graph->_input_tensors[0]);

    ComputeGraph *l1loss_graph = new L1LossGraph();


    Tensor *input = new Tensor({2, 1});

    Tensor *target = new Tensor({2, 1});


    Network mm_net(mm_graph, true, hipStreamDefault);
    mm_net.to(hipMemoryTypeDevice);
    mm_net._weight_tensors[0]->fill_data_random(-1.0, 1.0);
    mm_net.init({input}, "");


    Network l1loss(l1loss_graph, true, hipStreamDefault);
    l1loss.to(hipMemoryTypeDevice);


    for (int i = 0; i < 999; i++) {
        input->fill_data_random(-1.0, 1.0);
        target->_p_data[0] = input->_p_data[0] + input->_p_data[1];
        target->_p_data[1] = input->_p_data[0] - input->_p_data[1];
        std::vector<Tensor*> predict = mm_net.forward({input});
        std::vector<Tensor*> loss = l1loss.forward({predict[0], target});

        std::cout << "loss: " << loss[0] << std::endl;

        l1loss.backward();
        *mm_net.get_output_tensors()[0] = *l1loss._input_tensors[0];
        mm_net.backward();

        mm_net.update_weights(0.1);
    }

}