#include "transpose.h"

__managed__ int matrix[N][M];
__managed__ int gpu_result[M][N];
__managed__ int cpu_result[M][N];


int main() {
    for (int y = 0; y < N; ++y) {
        for (int x = 0; x < M; ++x) {
            matrix[y][x] = rand() % 1024;
        }
    }

    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_gpu);
    hipEventCreate(&stop_cpu);

    hipEventRecord(start);
    hipEventSynchronize(start);

    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((M + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);

    int times = 1;

    for (int i = 0; i < times; ++i) {
        gpu_matrix_transpose<<<grid, block>>>(matrix, gpu_result);
        // gpu_shared_matrix_transpose<<<grid, block>>>(matrix, gpu_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    cpu_matrix_transpose(matrix, cpu_result);
    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);

    float time_gpu, time_cpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);
    bool err_naive_gpu = checkResult(gpu_result, cpu_result);

    hipEvent_t start_s, stop_s;
    hipEventCreate(&start_s);
    hipEventCreate(&stop_s);
    hipEventRecord(start_s);
    for (int i = 0; i < times; ++i) {
        gpu_shared_matrix_transpose<<<grid, block>>>(matrix, gpu_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_s);
    bool err_shared_gpu = checkResult(gpu_result, cpu_result);

    hipEvent_t start_b, stop_b;
    hipEventCreate(&start_b);
    hipEventCreate(&stop_b);
    hipEventRecord(start_b);
    for (int i = 0; i < times; ++i) {
        gpu_shared_bank_matrix_transpose<<<grid, block>>>(matrix, gpu_result);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop_b);
    bool err_shared_bank_gpu = checkResult(gpu_result, cpu_result);

    float time_shared, time_shared_bank;
    hipEventElapsedTime(&time_shared, start_s, stop_s);
    hipEventElapsedTime(&time_shared_bank, start_b, stop_b);

    

    printf("Naive GPU:\n");
    printf("Result: %s\n", err_naive_gpu ? "Error" : "Pass");
    printf("CPU time: %.2f\t GPU time: %.2f\n", time_cpu, time_gpu/times);

    printf("Shared Memory:\n");
    printf("Result: %s\n", err_shared_gpu ? "Error" : "Pass");
    printf("CPU time: %.2f\t GPU time: %.2f\n", time_cpu, time_shared/times);

    printf("Shared Bank:\n");
    printf("Result: %s\n", err_shared_bank_gpu ? "Error" : "Pass");
    printf("CPU time: %.2f\t GPU time: %.2f\n", time_cpu, time_shared_bank/times);

    hipEventDestroy(start);
    hipEventDestroy(stop_gpu);
    hipEventDestroy(stop_cpu);
    hipEventDestroy(start_s);
    hipEventDestroy(start_b);
    hipEventDestroy(stop_s);
    hipEventDestroy(stop_b);

}

/*
 	nvcc main.cu transpose.cu -o main && ./main 
	nvcc main.cu transpose.cu -o main && /usr/local/NVIDIA-Nsight-Compute/ncu --set full -f -o transpose_profile_report ./main
*/